#include "hip/hip_runtime.h"
/* Image Histogram .cu
 * Device code.
 */

 #include <stdio.h>
 #include "histogram.h"
 
 #include <hip/hip_runtime.h>
 using namespace std; 
 
#include <iostream>
#include <cstdio>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <hip/hip_runtime.h>
#include <iomanip>

void histogram(const cv::Mat& input, cv::Mat& output);

__global__ void histogram_kernel(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep, unsigned int *bins_d) {
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int bdimx = blockDim.x;
    
    int by = blockIdx.y;
    int ty = threadIdx.y;
    int bdimy = blockDim.y;
    
    //2D Index of current thread
    const int xIndex = bx * bdimx + tx;
    const int yIndex = by * bdimy + ty;
	
    const int out_tid  = yIndex * grayWidthStep + (3*xIndex);
    const int color_tid = yIndex * colorWidthStep + (3 * xIndex);
    if((xIndex<width) && (yIndex<height)){

        int value = input[color_tid];		//Blue first
        atomicAdd(&bins_d[value], 1);
    }
    output[out_tid] = input[color_tid];
}
 
 ////////////////////////////////////////////////////////////////////////////////
void histogram(const cv::Mat& input, cv::Mat& output) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Calculate total number of bytes of input and output image
	const int colorBytes = input.step * input.rows;
	const int grayBytes = output.step * output.rows;

	unsigned char *input_d, *output_d;
	unsigned int *bins_d;
	unsigned int *kernel_bins;
	
	kernel_bins = (unsigned int *)malloc(256*sizeof(unsigned int));

	// Allocate device memory
	hipMalloc<unsigned char>(&input_d,colorBytes);
	hipMalloc<unsigned char>(&output_d,grayBytes);
	
	hipMalloc<unsigned int>(&bins_d,256*sizeof(unsigned int));
	//SAFE_CALL(hipMalloc(void**)(&bins_d,256*sizeof(int),"CUDA Malloc Failed");
	hipMemset(bins_d, 0, 256*sizeof(unsigned int));

	// Copy data from OpenCV input image to device memory
	hipMemcpy(input_d,input.ptr(),colorBytes,hipMemcpyHostToDevice);

	// Specify a reasonable block size
	const dim3 block(16,16);

	// Calculate grid size to cover the whole image
	const dim3 grid((input.cols + block.x - 1)/block.x, (input.rows + block.y - 1)/block.y);

	hipEventRecord(start);
	// Launch the color conversion kernel
	histogram_kernel<<<grid,block>>>(input_d,output_d,input.cols,input.rows,input.step,output.step, bins_d);
	
	hipEventRecord(stop);
	
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

    
	
	hipMemcpy(kernel_bins, bins_d, 256*sizeof(unsigned int), hipMemcpyDeviceToHost);
	//Synchronie host and device to ensure that transfer is finished
	hipDeviceSynchronize();
	
	
	for(int i = 0; i < 255; i++){
		printf(" %u", kernel_bins[i]);
	}
	
	cout << "Time taken by program is : "<< fixed << setprecision(10) << milliseconds;
    	cout << " milliseconds " << endl;
	
	
    	hipFree(input_d);

	// Synchronize to check for any kernel launch errors
	hipDeviceSynchronize();

	// Copy back data from destination device meory to OpenCV output image
	hipMemcpy(output.ptr(),output_d,grayBytes,hipMemcpyDeviceToHost);

	// Free the device memory
	hipFree(input_d);
	hipFree(output_d);
}
