#include "hip/hip_runtime.h"
/* Histogram Kernel with Privatization
 * Device code.
 */

 #include <stdio.h>
 #include "histogram.h"
 
 #include <hip/hip_runtime.h>
 using namespace std; 
 
#include <iostream>
#include <cstdio>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <hip/hip_runtime.h>
#include <iomanip>

void histogram_private(const cv::Mat& input, cv::Mat& output);
int histo_gold(const cv::Mat& input, int height, int width, unsigned int *gold_bins);

__global__ void histogram_private_kernel(unsigned char* input_1d, unsigned char* output, int width, int height, int inputWidthStep, int outputWidthStep, unsigned int *bins_d) {
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int bdimx = blockDim.x;
  
    __syncthreads();
    
    //2D Index of current thread
    const int xIndex = bx * bdimx + tx;
	
   // unsigned int i = bx*bdimx + tx;
    if(xIndex<width*height){
        uint32_t value = input_2d[xIndex];
        atomicAdd(&bins_d[bx*256 + value], 1);
    }
    if(bx > 0){
        __syncthreads();
        for(unsigned int bin_index = tx; bin_index < 256; bin_index+=bdimx){
            unsigned int bin_amount = bins_d[bx*256 + bin_index];
            if(bin_amount > 0){
                atomicAdd(&bins_d[bin_index], bin_amount);
            }
        }
    }

}
 
 ////////////////////////////////////////////////////////////////////////////////
void histogram_private(const cv::Mat& input, cv::Mat& output) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Calculate total number of bytes of input and output image
	const int colorBytes = input.step * input.rows;
	const int grayBytes = output.step * output.rows;

	unsigned char *input_d, *output_d, *input_1dd;
	unsigned char *input_1dh;
	unsigned int *bins_d;
	unsigned int *kernel_bins;
	unsigned int *gold_bins;
	
	// Specify a reasonable block size
	const dim3 block(1024, 1, 1);

	// Calculate grid size to cover the whole image
	const dim3 grid(((input.cols + block.x - 1)/block.x)*((input.rows + block.y - 1)/block.y), 1, 1);
	
	gold_bins = (unsigned int *)malloc(256*sizeof(unsigned int));
	kernel_bins = (unsigned int *)malloc(256*sizeof(unsigned int));
	
	input_1dh = (unsigned char *)malloc(colorBytes*sizeof(unsigned char));

	// Allocate device memory
	//hipMalloc<unsigned char>(&input_d,colorBytes);
	hipMalloc<unsigned char>(&output_d,grayBytes);
	hipMalloc<unsigned char>(&input_1dd,colorBytes);
	
	int gridSize = ((input.cols + block.x - 1)/block.x)*((input.rows + block.y - 1)/block.y);
	
	hipMalloc<unsigned int>(&bins_d,gridSize*sizeof(unsigned int));
	
	
	//SAFE_CALL(hipMalloc(void**)(&bins_d,256*sizeof(int),"CUDA Malloc Failed");
	hipMemset(bins_d, 0, 256*sizeof(int));
	
	hipMemset(input_1dd, 0, colorBytes*input.cols*sizeof(unsigned char));

	// Copy data from OpenCV input image to device memory
	//hipMemcpy(input_d,input.ptr(),colorBytes,hipMemcpyHostToDevice);
	
//	unsigned int m = 0;

  //      for (int j = 0; j < input.rows; ++j)
    //    {
        //	for (int i = 0; i < input.cols; ++i)
	///        {
	   // 		input_1d[m] = input.at<uchar>(j, i);
	    //		m++;
		//}
         //}



	unsigned int m = 0;

        for (int j = 0; j < input.rows; ++j)
        {
        	for (int i = 0; i < input.cols; ++i)
	        {
	    		input_1dh[m] = input.at<uchar>(j, i);
	    		m++;
		}
         }
	
	//// Copy data from OpenCV input image to device memory
	//hipMemcpy(input_1d,input.ptr(),colorBytes,hipMemcpyHostToDevice);
	hipMemcpy(input_1dd,input_1dh,colorBytes,hipMemcpyHostToDevice);
	
	//printf("%d ", (input.cols + block.x - 1)/block.x);
	//printf("%d ", (input.rows + block.y - 1)/block.y);


	hipEventRecord(start);
	// Launch the color conversion kernel
	histogram_private_kernel<<<grid,block>>>(input_1dd,output_d,input.cols,input.rows,input.step,output.step, bins_d);
	
	hipEventRecord(stop);
	
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

    
	
	hipMemcpy(kernel_bins, bins_d, 256*sizeof(unsigned int), hipMemcpyDeviceToHost);
	//Synchronie host and device to ensure that transfer is finished
	hipDeviceSynchronize();
	
	
	//for(int i = 0; i < 256; i++){
	//	printf(" K: %u", kernel_bins[i]);
	//	printf(" G: %u", gold_bins[i]);
	//}
	
	cout << "Time taken by program is : "<< fixed << setprecision(10) << milliseconds;
    	cout << " milliseconds " << endl;
    	
    	histo_gold(input, input.rows, input.cols, gold_bins);
    	cout << "\n";
    	
    	for(int i = 0; i < 256; i++){
    		printf(" K: ");
		printf("%u", kernel_bins[i]);
		printf(" G: ");
		printf("%u", gold_bins[i]);
	}
    	
	int passed=1;
        for (int i=0; i < 256; i++){
        	if (gold_bins[i] != kernel_bins[i]){
	       	       passed = 0;
	       	       break;
	        }
         }
         (passed) ? printf("\n    Test PASSED\n") : printf("\n    Test FAILED\n");
	
	free(input_1dh);
    	hipFree(input_1dd);

	// Synchronize to check for any kernel launch errors
	hipDeviceSynchronize();

	// Copy back data from destination device meory to OpenCV output image
	hipMemcpy(output.ptr(),output_d,grayBytes,hipMemcpyDeviceToHost);

	free(kernel_bins);
	free(gold_bins);
	// Free the device memory
	//hipFree(input_d);
	hipFree(output_d);
}



int histo_gold(const cv::Mat& input, int height, int width, unsigned int *gold_bins)
{

    // Zero out all the bins
    memset(gold_bins, 0, 256*sizeof(unsigned int));

    for (int j = 0; j < height; ++j)
    {
        for (int i = 0; i < width; ++i)
        {
            unsigned int value = input.at<uchar>(j, i);

            gold_bins[value]++;
            //unsigned int *p = (unsigned int*)gold_bins;
            //++p[value];
        }
    }

    return 0;
}


